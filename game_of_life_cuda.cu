#include "hip/hip_runtime.h"
/**
 * Conway's Game of Life using Cuda
 * 
 * This version runs in parallel on a GPU using Cuda. Compile with:
 * 	   gcc -Wall -O3 -march=native -c helpers.c
 *     nvcc -arch=sm_20 -O3 game_of_life_cuda.cu helpers.o -o game_of_life_cuda -lm
 * And run with:
 * 	   ./game_of_life_cuda [-n num-of-iterations] [-r random-seed] [-s world_size]
 */

 #include<stdio.h>
 #include<stdlib.h>
 #include <unistd.h>
 #include <inttypes.h>
 #include <time.h>
 #include <string.h>
 #include <hip/hip_runtime.h>


 #define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
      exit(1);                                                            \
   }                                                                      \
}

void print_world(size_t* grid, size_t world_size) {
	for (size_t i = 0; i < world_size; i++) {
		for (size_t j = 0; j < world_size; j++) {
			printf("%lu, ", grid[i*world_size+j]);
		}
		printf("\n");
	}
    printf("\n");
}

__device__
int get_num_neighbors(size_t* grid, const size_t i, const size_t world_size, const size_t grid_size) {
	const int half_neighbors = 4;
    size_t neighbor_count = 0;
    int before_positions[half_neighbors] = {i - world_size - 1, i - world_size, i - world_size + 1, i - 1};
    int after_positions[half_neighbors] = {i + 1, i + world_size - 1, i + world_size, i + world_size + 1};
    
    for (size_t j = 0; j < half_neighbors; j++) {
        if (before_positions[j] >= 0) {
            if (grid[before_positions[j]]) neighbor_count++;
        }
        if (after_positions[j] < grid_size) {
            if (grid[after_positions[j]]) neighbor_count++;
        }
    }
 
   return neighbor_count;
 }

 /**
 * Make the current position array the next position array
 */
 __device__
void swap(size_t** grid, size_t** grid_next) {
    size_t* temp = *grid;
    *grid = *grid_next;
    *grid_next = temp;
}

__global__
void simulate(uint8_t* grid, size_t* grid_next, const size_t world_size, const size_t iterations) {
	const int i = threadIdx.x, grid_size = blockDim.x;
    for (size_t step = 0; step < iterations; step++) {
		for (j += blockDim.x) {
			// i+j*blockDim.x;
			size_t num_neighbors = get_num_neighbors(grid, i, world_size, grid_size);
			if (grid[i]) { grid_next[i] = num_neighbors <= 1 || num_neighbors > 3 ? 0 : 1; }
			else { grid_next[i] = num_neighbors == 3 ? 1 : 0; }
		}

		swap(&grid, &grid_next);
		__syncthreads();
  	}
}


int main(int argc, char* const argv[]) {
	size_t iterations = 1;
	size_t random_seed = time(NULL);
    size_t world_size = 20;
 
     // parse command line arguments
     int opt;
     while ((opt = getopt(argc, argv, "n:r:s:")) != -1) {
         char* end;
         switch (opt) {
         case 'n': iterations = strtoumax(optarg, &end, 10); break;
         case 'r': random_seed = strtoul(optarg, &end, 10); break;
         case 's': world_size = strtoumax(optarg, &end, 10); break;
         default:
             fprintf(stderr, "usage: %s [-n num-iterations] [-r random-seed] [-s world-size] input output\n", argv[0]);
             return 1;
         }
     }
     if (optind + 2 < argc || iterations == 0) {
         fprintf(stderr, "usage: %s [-n num-iterations] [-r random-seed] [-s world-size] input output\n", argv[0]);
         return 1;
     }
 
	const size_t grid_size = world_size * world_size;
	const size_t grid_bytes = grid_size*sizeof(size_t);
	size_t* h_grid = (size_t*) malloc(grid_bytes);
	size_t* h_grid_next = (size_t*) malloc(grid_bytes);
	memset(h_grid, 0, grid_bytes);

	size_t *d_grid, *d_grid_next;
 
     // Use this for random data
    //  srand(random_seed);
    //  for (size_t i = 0; i < grid_size; i++) h_grid[i] = rand() & 1;
 
     // Use this for the same simulation as the original
    for (size_t i = 145; i < 155; i++) h_grid[i] = 1;
    h_grid[282] = 1;
    h_grid[283] = 1;
    h_grid[284] = 1;
    h_grid[285] = 1;
    h_grid[301] = 1;
    h_grid[305] = 1;
    h_grid[325] = 1;
    h_grid[341] = 1;
    h_grid[344] = 1;

    CHECK(hipMalloc(&d_grid, grid_bytes));
    CHECK(hipMalloc(&d_grid_next, grid_bytes));

    CHECK(hipMemcpy(d_grid, h_grid, grid_bytes, hipMemcpyHostToDevice));
 
    simulate<<<1, grid_size>>>(d_grid, d_grid_next, world_size, iterations);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_grid_next, d_grid_next, grid_bytes, hipMemcpyDeviceToHost));

    print_world(h_grid_next, world_size);
    free(h_grid); free(h_grid_next);
    CHECK(hipFree(d_grid)); CHECK(hipFree(d_grid_next));
    return 0;
}
 